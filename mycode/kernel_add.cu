
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void kernel_add(const int*a, const int *b, int*c){
    *c = *a + *b;
}

int main(){
    int a = 1, b = 2, c;
    int *d_a, *d_b, *d_c;
    
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    kernel_add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("c : %d\n",c);
}