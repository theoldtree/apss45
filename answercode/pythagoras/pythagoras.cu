
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *presult) {
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *presult = 1;
  else
    *presult = 0;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int result = 0;

  // TODO: 1. allocate device memory
  int *d_a, *d_b, *d_c, *d_result;
  CHECK_CUDA(hipMalloc(&d_a, sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_b, sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_c, sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_result, sizeof(int)));

  // TODO: 2. copy data to device
  CHECK_CUDA(hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice));

  // TODO: 3. launch kernel
  pythagoras<<<1,1>>>(d_a,d_b,d_c,d_result);
  CHECK_CUDA(hipGetLastError());

  // TODO: 4. copy result back to host
  CHECK_CUDA(hipMemcpy(&result,d_result,sizeof(int),hipMemcpyDeviceToHost));

  if (result) printf("YES\n");
  else printf("NO\n");

  return 0;
}
